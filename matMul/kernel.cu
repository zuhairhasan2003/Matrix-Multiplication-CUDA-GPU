#include "hip/hip_runtime.h"
﻿
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void MatMul(int* mat1, int* mat2, int* ansMat, int row1, int col1row2, int col2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    if (row < row1 && col < col2)
    {
        int sum = 0;
        for (int k = 0; k < col1row2; k++)
        {
            sum += mat1[row * col1row2 + k] * mat2[col2 * k + col];
        }
        ansMat[row * col2 + col] = sum;
    }
}

void InitMatrix(int*, int, int);
void DisplayMatrix(int*, int, int);

int main()
{
    int row1 = 500;
    int col1 = 900;
    int* mat1 = (int*)malloc((row1 * col1) * sizeof(int));
    InitMatrix(mat1, row1, col1);


    int row2 = 900;
    int col2 = 500;
    int* mat2 = (int*)malloc((row2 * col2) * sizeof(int));
    InitMatrix(mat2, row2, col2);

    int ansMatRow = row1;
    int ansMatCol = col2;
    int* ansMat = (int*)malloc((ansMatCol * ansMatRow) * sizeof(int));

    int* d_mat1;
    int* d_mat2;
    int* d_ansMat;

    hipMalloc((void**)&d_mat1, row1 * col1 * sizeof(int));
    hipMalloc((void**)&d_mat2, row2 * col2 * sizeof(int));
    hipMalloc((void**)&d_ansMat, ansMatCol * ansMatRow * sizeof(int));

    hipMemcpy(d_mat1, mat1, (row1 * col1 * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, mat2, (row2 * col2 * sizeof(int)), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim(col2 / blockDim.x + 1, row1 / blockDim.y + 1);

    MatMul << <gridDim, blockDim >> > (d_mat1, d_mat2, d_ansMat, row1, col1, col2);
    hipDeviceSynchronize();

    hipMemcpy(ansMat, d_ansMat, (row1 * col2 * sizeof(int)), hipMemcpyDeviceToHost);

    //DisplayMatrix(mat1, row1, col1);
    //printf("\n");
    //DisplayMatrix(mat2, row2, col2);
    //printf("\n");
    //DisplayMatrix(ansMat, ansMatRow, ansMatCol);


    return 0;
}

void InitMatrix(int* mat, int row, int col)
{
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            mat[(i * col) + j] = rand() % 5;
        }
    }
}

void DisplayMatrix(int* mat, int row, int col)
{
    for (int i = 0; i < row; i++)
    {
        printf("[  ");
        for (int j = 0; j < col; j++)
        {
            printf("%d   ", mat[(i * col) + j]);
        }
        printf("]\n");
    }
}
